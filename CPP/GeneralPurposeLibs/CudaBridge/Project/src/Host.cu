#include "hip/hip_runtime.h"

#include "../include/Host.cuh"

__host__ void RA::Host::PrintDeviceStats()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);


    if (deviceCount == 0)
        printf("No CUDA support device found\n\n");
    else if (deviceCount > 1)
        printf("You have SLi Running!!\n\n");
    else
        printf("You have one video card running\n\n");

    printf("Number of devices:                               %d\n", deviceCount);

    const char* Line = "\n---------------------------------------------------------------------------\n";
    for (int devNo = 0; devNo < deviceCount; devNo++) {

        hipDeviceProp_t iProp;
        printf(Line);
        hipGetDeviceProperties(&iProp, devNo);
        printf("Device %d Model:                                 %s\n", devNo, iProp.name);
        printf("  Number of multiprocessors:                     %d\n", iProp.multiProcessorCount);
        printf("  clock rate:                                    %d\n", iProp.clockRate);
        printf("  Compute capability:                            %d.%d\n", iProp.major, iProp.minor);
        printf("  Total amount of global memory:                 %4.2f KB\n", iProp.totalGlobalMem / 1024.0);
        printf("  Total amount of constant memory:               %4.2f KB\n", iProp.totalConstMem / 1024.0);
        printf("  Total amount of shared memory per block:       %4.2f KB\n", iProp.sharedMemPerBlock / 1024.0);
        printf("  Total amount of shared memory per MP:          %4.2f KB\n", iProp.sharedMemPerMultiprocessor / 1024.0);
        printf("  Total number of registers available per block: %d\n", iProp.regsPerBlock);
        printf("  Warp size:                                     %d\n", iProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n", iProp.maxThreadsPerBlock);
        printf("  Maximum number of threads per multiprocessor:  %d\n", iProp.maxThreadsPerMultiProcessor);
        printf("  Maximum number of warps per multiprocessor:    %d\n", iProp.maxThreadsPerMultiProcessor / 32);

        printf("  Maximum Grid size:                            (%d,%d,%d)\n",
            iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);

        printf("  Maximum block dimension:                      (%d,%d,%d)\n",
            iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
    }
    printf(Line);
}
