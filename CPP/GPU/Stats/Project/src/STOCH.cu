#include "hip/hip_runtime.h"
﻿// Copyright via Apache v2 Licence [2023][Joel Leagues aka Scourge]
#if UsingMSVC
#include "STOCH.h"
#else
#include "STOCH.cuh"
#endif

RA::STOCH::STOCH(
    const double* FvValues,
    const xint    FnLogicalSize,
    const xint* FnStorageSizePtr,
    const xint* FnInsertIdxPtr)
    :
    MvValues(FvValues),
    MnLogicalSize(FnLogicalSize),
    MnStorageSizePtr(FnStorageSizePtr),
    MnInsertIdxPtr(FnInsertIdxPtr)
{
    Begin();
    // commented out because you can use RA::Stats::Construct
    //if (!*MnStorageSizePtr)
    //    ThrowIt("STOCH needs storage values");
    Rescue();
}

DXF void RA::STOCH::CopyStats(const RA::STOCH& Other)
{
    MnLogicalSize = Other.MnLogicalSize;
    MnSmallest = Other.MnSmallest;
    MnBiggest = Other.MnBiggest;
    MnSTOCH = Other.MnSTOCH;
}

DXF void RA::STOCH::Update()
{
    if (MnStorageSizePtr == nullptr || *MnStorageSizePtr == 0)
    {
        printf(RED "RSI needs storage to work\n" WHITE);
        return;
    }

    MnBiggest = -DBL_MAX; // min is big   until proven otherwise
    MnSmallest = +DBL_MAX; // max is small until proven otherwise

    const auto& LnStart = *The.MnInsertIdxPtr;
    const auto& LnStorage = *The.MnStorageSizePtr;
    const auto& LnLogic = The.MnLogicalSize;

    xint Idx = LnStart;
    for (xint i = LnStart; i < LnStart + LnLogic; i++)
    {
        // note: the first value will remove the possibility of DBL_MAX/DBL_MIN
        if (MvValues[Idx] > MnBiggest)
            MnBiggest = MvValues[Idx];
        if (MvValues[Idx] < MnSmallest)
            MnSmallest = MvValues[Idx];

        Idx = (Idx == 0) ? LnStorage - 1 : Idx - 1;
    }

    if (BxNoEntry() || (MnBiggest - MnSmallest == 0))
    {
        MnSTOCH = 50;
        return;
    }

    const auto& LnCurrent = MvValues[LnStart];
    MnSTOCH = 100 * ((LnCurrent - MnSmallest) / (MnBiggest - MnSmallest));
    //cout << "stoch val: " << MvValues[LnStart] << " : " << MnSTOCH << endl;
}

DXF void RA::STOCH::Update(const double FnValue)
{
    if (MnBiggest < FnValue)
        MnBiggest = FnValue;
    if (MnSmallest > FnValue)
        MnSmallest = FnValue;
    MnSTOCH = 100 * ((FnValue - MnSmallest) / (MnBiggest - MnSmallest));
}

DXF void RA::STOCH::SetLogicalSize(const xint FnLogicalSize)
{
    MnLogicalSize = (FnLogicalSize <= *MnStorageSizePtr) ? FnLogicalSize : *MnStorageSizePtr;
}

DXF void RA::STOCH::SetDefaultValues(const double FnDefaualt)
{
    MnBiggest = FnDefaualt;
    MnSmallest = FnDefaualt;
    MnSTOCH = 50;
}

