﻿// Copyright via Apache v2 Licence [2023][Joel Leagues aka Scourge]
#if UsingMSVC
#include "StatsCPU.h"
#else
#include "StatsCPU.cuh"
#endif

RA::StatsCPU::StatsCPU()
{
    MeHardware = EHardware::CPU;
}

RA::StatsCPU::StatsCPU(const StatsCPU& Other)
{
    Begin();
    The = Other;
    Rescue();
}

RA::StatsCPU::StatsCPU(StatsCPU&& Other) noexcept
{
    The = std::move(Other);
}

void RA::StatsCPU::operator=(const StatsCPU& Other)
{
    Begin();
    MeHardware = Other.MeHardware;
    if (!!Other.MvValues && Other.MnStorageSize)
    {
        Allocate(Other.MnStorageSize, 1);
        //memcpy(MvValues, Other.MvValues, MnStorageSize);
        MbHadFirstInsert = Other.MbHadFirstInsert;
        MnInsertIdx = Other.MnInsertIdx;
    }
    else
    {
        MnInsertIdx = 0;
        MbHadFirstInsert = false;
    }

    The.ConstructHardware(MeHardware, Other.MnStorageSize, Other.MvOptions);
    The.SetJoinerySize(Other.MoJoinery.MnSize);

    if(MnStorageSize)
        The.SetSlippageSize(Other.MoSlippage.MnSlipSize);

    if (Other.MoAvgPtr)         MoAvgPtr->CopyStats(*Other.MoAvgPtr);
    if (Other.MoSTOCHPtr)       MoSTOCHPtr->CopyStats(*Other.MoSTOCHPtr);
    if (Other.MoRSIPtr)         MoRSIPtr->CopyStats(*Other.MoRSIPtr);
    if (Other.MoStandardDeviationPtr)     MoStandardDeviationPtr->CopyStats(*Other.MoStandardDeviationPtr);
    if (Other.MoMeanAbsoluteDeviationPtr) MoMeanAbsoluteDeviationPtr->CopyStats(*Other.MoMeanAbsoluteDeviationPtr);

    The.SetDeviceJoinery();

    Rescue();
}

void RA::StatsCPU::operator=(StatsCPU&& Other) noexcept
{
    Other.MbDelete = false;
    MeHardware = Other.MeHardware;
    Clear();
    if (!!Other.MvValues && Other.MnStorageSize)
    {
        MvValues = Other.MvValues;
        MnInsertIdx = Other.MnInsertIdx;
    }
    else
    {
        MnInsertIdx = 0;
        MbHadFirstInsert = false;
    }

    MeHardware    = Other.MeHardware;
    MnStorageSize = Other.MnStorageSize;
    MnInsertIdx   = 0;

    if (Other.GetJoinerySize())
    {
        SetJoinerySize(Other.GetJoinerySize());
        for (xint i = 0; i < Other.GetJoinerySize(); i++)
            MoJoinery.MvValues[i] = Other.MoJoinery.MvValues[i];
    }

    if (Other.MoSlippage.MnSlipSize)
    {
        SetSlippageSize(Other.MoSlippage.MnSlipSize);
        for (xint i = 0; i < Other.MoSlippage.MnDataLeng; i++)
            MoSlippage.MvNums[i] = Other.MoSlippage.MvNums[i];
    }

    MbHadFirstInsert = Other.MbHadFirstInsert;
    MvOptions        = Other.MvOptions;

    MoAvgPtr        = Other.MoAvgPtr;
    MoRSIPtr        = Other.MoRSIPtr;
    MoSTOCHPtr      = Other.MoSTOCHPtr;
    MoStandardDeviationPtr     = Other.MoStandardDeviationPtr;
    MoMeanAbsoluteDeviationPtr = Other.MoMeanAbsoluteDeviationPtr;

    The.SetDeviceJoinery();
}

RA::StatsCPU::StatsCPU(
    const xint FnStorageSize, 
    const xvector<EStatOpt> FvOptions,
    const double FnDefaultVal)
    : RA::Stats(RA::EHardware::CPU, FnStorageSize, FvOptions, FnDefaultVal)
{
}

// --------------------------------------------------------

DHF RA::AVG& RA::StatsCPU::GetObjAVG()
{
    Begin();
    if (!MoAvgPtr)
        ThrowIt("MoAvgPtr is Null");
    return *MoAvgPtr;
    Rescue();
}

DHF RA::STOCH& RA::StatsCPU::GetObjSTOCH()
{
    Begin();
    if (!MoSTOCHPtr)
        ThrowIt("MoSTOCHPtr is Null");
    return *MoSTOCHPtr;
    Rescue();
}

DHF RA::RSI& RA::StatsCPU::GetObjRSI()
{
    Begin();
    if (!MoRSIPtr)
        ThrowIt("MoRSIPtr is Null");
    return *MoRSIPtr;
    Rescue();
}

DHF RA::Deviation& RA::StatsCPU::GetObjStandardDeviation()
{
    Begin();
    if (!MoStandardDeviationPtr)
        ThrowIt("MoStandardDeviationPtr is Null");
    return *MoStandardDeviationPtr;
    Rescue();
}


DHF RA::Deviation& RA::StatsCPU::GetObjMeanAbsoluteDeviation()
{
    Begin();
    if (!MoMeanAbsoluteDeviationPtr)
        ThrowIt("MoMeanAbsoluteDeviationPtr is Null");
    return *MoMeanAbsoluteDeviationPtr;
    Rescue();
}

// --------------------------------------------------------

DHF const RA::AVG& RA::StatsCPU::GetObjAVG() const
{
    Begin();
    if (!MoAvgPtr)
        ThrowIt("MoAvgPtr is Null");
    return *MoAvgPtr;
    Rescue();
}

DHF const RA::STOCH& RA::StatsCPU::GetObjSTOCH() const
{
    Begin();
    if (!MoSTOCHPtr)
        ThrowIt("MoSTOCHPtr is Null");
    return *MoSTOCHPtr;
    Rescue();
}

DHF const RA::RSI& RA::StatsCPU::GetObjRSI() const
{
    Begin();
    if (!MoRSIPtr)
        ThrowIt("MoRSIPtr is Null");
    return *MoRSIPtr;
    Rescue();
}

DHF const RA::Deviation& RA::StatsCPU::GetObjStandardDeviation() const
{
    Begin();
    if (!MoStandardDeviationPtr)
        ThrowIt("MoStandardDeviationPtr is Null");
    return *MoStandardDeviationPtr;
    Rescue();
}


DHF const RA::Deviation& RA::StatsCPU::GetObjMeanAbsoluteDeviation() const
{
    Begin();
    if (!MoMeanAbsoluteDeviationPtr)
        ThrowIt("MoMeanAbsoluteDeviationPtr is Null");
    return *MoMeanAbsoluteDeviationPtr;
    Rescue();
}

// --------------------------------------------------------

DHF const RA::AVG& RA::StatsCPU::AVG() const
{
    Begin();
    if (!MoAvgPtr)
        ThrowIt("MoAvgPtr is Null");
    return *MoAvgPtr;
    Rescue();
}

DHF const RA::STOCH& RA::StatsCPU::STOCH() const
{
    Begin();
    if (!MoSTOCHPtr)
        ThrowIt("MoSTOCHPtr is Null");
    return *MoSTOCHPtr;
    Rescue();
}

DHF const RA::RSI& RA::StatsCPU::RSI() const
{
    Begin();
    if (!MoRSIPtr)
        ThrowIt("MoRSIPtr is Null");
    return *MoRSIPtr;
    Rescue();
}

DHF const RA::Deviation& RA::StatsCPU::SD() const
{
    Begin();
    if (!MoStandardDeviationPtr)
        ThrowIt("MoStandardDeviationPtr is Null");
    return *MoStandardDeviationPtr;
    Rescue();
}

DHF const RA::Deviation& RA::StatsCPU::MAD() const
{
    Begin();
    if (!MoMeanAbsoluteDeviationPtr)
        ThrowIt("MoMeanAbsoluteDeviationPtr is Null");
    return *MoMeanAbsoluteDeviationPtr;
    Rescue();
}

// --------------------------------------------------------

DHF RA::Deviation& RA::StatsCPU::SD()
{
    Begin();
    if (!MoStandardDeviationPtr)
        ThrowIt("MoStandardDeviationPtr is Null");
    return *MoStandardDeviationPtr;
    Rescue();
}

DHF RA::Deviation& RA::StatsCPU::MAD()
{
    Begin();
    if (!MoMeanAbsoluteDeviationPtr)
        ThrowIt("MoMeanAbsoluteDeviationPtr is Null");
    return *MoMeanAbsoluteDeviationPtr;
    Rescue();
}